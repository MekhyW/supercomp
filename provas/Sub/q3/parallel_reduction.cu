#include <iostream>
#include <vector>
#include <cstdlib>
#include <ctime>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>

int main() {
    std::srand(std::time(nullptr));
    const size_t N = 200000;
    thrust::host_vector<int> h_vec(N);
    for (size_t i = 0; i < N; ++i) {
        h_vec[i] = std::rand() % 100;
    }
    thrust::device_vector<int> d_vec = h_vec;
    int sum = thrust::reduce(d_vec.begin(), d_vec.end(), 0, thrust::plus<int>());
    std::cout << "Sum of elements: " << sum << std::endl;
    return 0;
}
